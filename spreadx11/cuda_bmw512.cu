#if 1

#include <hip/hip_runtime.h>



#include <stdint.h>
#include <stdio.h>
#include <memory.h>

#define TPB 128

static __device__ uint32_t cuda_swab32(uint32_t x)
{
	return __byte_perm(x, 0, 0x0123);
}

// das Hi Word in einem 64 Bit Typen ersetzen
static __device__ __forceinline__ unsigned long long REPLACE_HIWORD(const unsigned long long &x, const uint32_t &y) {
	return (x & 0xFFFFFFFFULL) | (((unsigned long long)y) << 32ULL);
}

// die Message it Padding zur Berechnung auf der GPU
__constant__ uint64_t c_PaddedMessage80[16]; // padded message (80 bytes + padding)

#define SPH_C64(x)    ((uint64_t)(x ## ULL))

__forceinline__ __device__ uint64_t ROTL64S(const uint64_t value, const int offset) {
    uint64_t result;
    asm("{\n\t"
    " .reg .u32 tl,th,vl,vh; \n\t"
    "mov.b64 {tl,th},%1; \n\t"
    "shf.l.wrap.b32 vh,tl,th,%2; \n\t"
    "shf.l.wrap.b32 vl,th,tl,%2; \n\t"
    "mov.b64 %0,{vl,vh}; \n\t"
    "}"
    : "=l"(result) : "l"(value) , "r"(offset));
    return  result;
}

__forceinline__ __device__ uint64_t ROTL64_32(const uint64_t value) {
    uint64_t result;
    asm("{\n\t"
    " .reg .u32 tl,th; \n\t"
    "mov.b64 {tl,th},%1; \n\t"
    "mov.b64 %0,{th,tl}; \n\t"
    "}"
    : "=l"(result) : "l"(value));
    return  result;
}

__forceinline__ __device__ uint64_t ROTL64B(const uint64_t value, const int offset) {
    uint64_t result;
    asm("{\n\t"
    " .reg .u32 tl,th,vl,vh; \n\t"
    "mov.b64 {tl,th},%1; \n\t"
    "shf.l.wrap.b32 vl,tl,th,%2; \n\t"
    "shf.l.wrap.b32 vh,th,tl,%2; \n\t"
    "mov.b64 %0,{vl,vh}; \n\t"
    "}"
    : "=l"(result) : "l"(value) , "r"(offset));
    return  result;
}

static __constant__ uint64_t d_constMem[16];
static uint64_t h_constMem[16] = {
	SPH_C64(0x8081828384858687),
    SPH_C64(0x88898A8B8C8D8E8F),
    SPH_C64(0x9091929394959697),
    SPH_C64(0x98999A9B9C9D9E9F),
    SPH_C64(0xA0A1A2A3A4A5A6A7),
    SPH_C64(0xA8A9AAABACADAEAF),
    SPH_C64(0xB0B1B2B3B4B5B6B7),
    SPH_C64(0xB8B9BABBBCBDBEBF),
    SPH_C64(0xC0C1C2C3C4C5C6C7),
    SPH_C64(0xC8C9CACBCCCDCECF),
    SPH_C64(0xD0D1D2D3D4D5D6D7),
    SPH_C64(0xD8D9DADBDCDDDEDF),
    SPH_C64(0xE0E1E2E3E4E5E6E7),
    SPH_C64(0xE8E9EAEBECEDEEEF),
    SPH_C64(0xF0F1F2F3F4F5F6F7),
    SPH_C64(0xF8F9FAFBFCFDFEFF)
};

#define Kb(j)   ((uint64_t)(j) * 0x0555555555555555ull)
#define Kc(j)   ((uint64_t)(j) + 0xaaaaaaaaaaaaaaa0ull)

static __constant__ uint64_t d_x55[16];
static const uint64_t h_x55[16] = {
	Kb(16), Kb(17), Kb(18), Kb(19), Kb(20), Kb(21), Kb(22), Kb(23),
    Kb(24), Kb(25), Kb(26), Kb(27), Kb(28), Kb(29), Kb(30), Kb(31)
};

static __constant__ uint64_t d_final[16];
static const uint64_t h_final[16] = {
    Kc(0), Kc(1), Kc(2), Kc(3), Kc(4), Kc(5), Kc(6), Kc(7),
    Kc(8), Kc(9), Kc(10), Kc(11), Kc(12), Kc(13), Kc(14), Kc(15)
};

#define SHL(x, n)            ((x) << (n))
#define SHR(x, n)            ((x) >> (n))

#define CONST_EXP2    q[i+0] + ROTL64S(q[i+1], 5)  + q[i+2] + ROTL64S(q[i+3], 11) + \
                    q[i+4] + ROTL64S(q[i+5], 27) + q[i+6] + ROTL64_32(q[i+7]) + \
                    q[i+8] + ROTL64B(q[i+9], 37) + q[i+10] + ROTL64B(q[i+11], 43) + \
                    q[i+12] + ROTL64B(q[i+13], 53) + (SHR(q[i+14],1) ^ q[i+14]) + (SHR(q[i+15],2) ^ q[i+15])

__device__ void Compression512(uint64_t *msg, uint64_t *hash)
{
    // Compression ref. implementation
    uint64_t tmp[16];
    uint64_t q[32];

    tmp[0] = (msg[ 5] ^ hash[ 5]) - (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]) + (msg[14] ^ hash[14]);
    tmp[1] = (msg[ 6] ^ hash[ 6]) - (msg[ 8] ^ hash[ 8]) + (msg[11] ^ hash[11]) + (msg[14] ^ hash[14]) - (msg[15] ^ hash[15]);
    tmp[2] = (msg[ 0] ^ hash[ 0]) + (msg[ 7] ^ hash[ 7]) + (msg[ 9] ^ hash[ 9]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    tmp[3] = (msg[ 0] ^ hash[ 0]) - (msg[ 1] ^ hash[ 1]) + (msg[ 8] ^ hash[ 8]) - (msg[10] ^ hash[10]) + (msg[13] ^ hash[13]);
    tmp[4] = (msg[ 1] ^ hash[ 1]) + (msg[ 2] ^ hash[ 2]) + (msg[ 9] ^ hash[ 9]) - (msg[11] ^ hash[11]) - (msg[14] ^ hash[14]);
    tmp[5] = (msg[ 3] ^ hash[ 3]) - (msg[ 2] ^ hash[ 2]) + (msg[10] ^ hash[10]) - (msg[12] ^ hash[12]) + (msg[15] ^ hash[15]);
    tmp[6] = (msg[ 4] ^ hash[ 4]) - (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) - (msg[11] ^ hash[11]) + (msg[13] ^ hash[13]);
    tmp[7] = (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 5] ^ hash[ 5]) - (msg[12] ^ hash[12]) - (msg[14] ^ hash[14]);
    tmp[8] = (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) - (msg[ 6] ^ hash[ 6]) + (msg[13] ^ hash[13]) - (msg[15] ^ hash[15]);
    tmp[9] = (msg[ 0] ^ hash[ 0]) - (msg[ 3] ^ hash[ 3]) + (msg[ 6] ^ hash[ 6]) - (msg[ 7] ^ hash[ 7]) + (msg[14] ^ hash[14]);
    tmp[10] = (msg[ 8] ^ hash[ 8]) - (msg[ 1] ^ hash[ 1]) - (msg[ 4] ^ hash[ 4]) - (msg[ 7] ^ hash[ 7]) + (msg[15] ^ hash[15]);
    tmp[11] = (msg[ 8] ^ hash[ 8]) - (msg[ 0] ^ hash[ 0]) - (msg[ 2] ^ hash[ 2]) - (msg[ 5] ^ hash[ 5]) + (msg[ 9] ^ hash[ 9]);
    tmp[12] = (msg[ 1] ^ hash[ 1]) + (msg[ 3] ^ hash[ 3]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[10] ^ hash[10]);
    tmp[13] = (msg[ 2] ^ hash[ 2]) + (msg[ 4] ^ hash[ 4]) + (msg[ 7] ^ hash[ 7]) + (msg[10] ^ hash[10]) + (msg[11] ^ hash[11]);
    tmp[14] = (msg[ 3] ^ hash[ 3]) - (msg[ 5] ^ hash[ 5]) + (msg[ 8] ^ hash[ 8]) - (msg[11] ^ hash[11]) - (msg[12] ^ hash[12]);
    tmp[15] = (msg[12] ^ hash[12]) - (msg[ 4] ^ hash[ 4]) - (msg[ 6] ^ hash[ 6]) - (msg[ 9] ^ hash[ 9]) + (msg[13] ^ hash[13]);
    
    q[0] = (SHR(tmp[0], 1) ^ SHL(tmp[0], 3) ^ ROTL64S(tmp[0],  4) ^ ROTL64B(tmp[0], 37)) + hash[1];
    q[1] = (SHR(tmp[1], 1) ^ SHL(tmp[1], 2) ^ ROTL64S(tmp[1], 13) ^ ROTL64B(tmp[1], 43)) + hash[2];
    q[2] = (SHR(tmp[2], 2) ^ SHL(tmp[2], 1) ^ ROTL64S(tmp[2], 19) ^ ROTL64B(tmp[2], 53)) + hash[3];
    q[3] = (SHR(tmp[3], 2) ^ SHL(tmp[3], 2) ^ ROTL64S(tmp[3], 28) ^ ROTL64B(tmp[3], 59)) + hash[4];
    q[4] = (SHR(tmp[4], 1) ^ tmp[4]) + hash[5];
    q[5] = (SHR(tmp[5], 1) ^ SHL(tmp[5], 3) ^ ROTL64S(tmp[5],  4) ^ ROTL64B(tmp[5], 37)) + hash[6];
    q[6] = (SHR(tmp[6], 1) ^ SHL(tmp[6], 2) ^ ROTL64S(tmp[6], 13) ^ ROTL64B(tmp[6], 43)) + hash[7];
    q[7] = (SHR(tmp[7], 2) ^ SHL(tmp[7], 1) ^ ROTL64S(tmp[7], 19) ^ ROTL64B(tmp[7], 53)) + hash[8];
    q[8] = (SHR(tmp[8], 2) ^ SHL(tmp[8], 2) ^ ROTL64S(tmp[8], 28) ^ ROTL64B(tmp[8], 59)) + hash[9];
    q[9] = (SHR(tmp[9], 1) ^ tmp[9]) + hash[10];
    q[10] = (SHR(tmp[10], 1) ^ SHL(tmp[10], 3) ^ ROTL64S(tmp[10],  4) ^ ROTL64B(tmp[10], 37)) + hash[11];
    q[11] = (SHR(tmp[11], 1) ^ SHL(tmp[11], 2) ^ ROTL64S(tmp[11], 13) ^ ROTL64B(tmp[11], 43)) + hash[12];
    q[12] = (SHR(tmp[12], 2) ^ SHL(tmp[12], 1) ^ ROTL64S(tmp[12], 19) ^ ROTL64B(tmp[12], 53)) + hash[13];
    q[13] = (SHR(tmp[13], 2) ^ SHL(tmp[13], 2) ^ ROTL64S(tmp[13], 28) ^ ROTL64B(tmp[13], 59)) + hash[14];
    q[14] = (SHR(tmp[14], 1) ^ tmp[14]) + hash[15];
    q[15] = (SHR(tmp[15], 1) ^ SHL(tmp[15], 3) ^ ROTL64S(tmp[15], 4) ^ ROTL64B(tmp[15], 37)) + hash[0];

    // Expand 1
    for(int i=0;i<2;i++)
    {
        q[i+16] =
        (SHR(q[i], 1) ^ SHL(q[i], 2) ^ ROTL64S(q[i], 13) ^ ROTL64B(q[i], 43)) +
        (SHR(q[i+1], 2) ^ SHL(q[i+1], 1) ^ ROTL64S(q[i+1], 19) ^ ROTL64B(q[i+1], 53)) +
        (SHR(q[i+2], 2) ^ SHL(q[i+2], 2) ^ ROTL64S(q[i+2], 28) ^ ROTL64B(q[i+2], 59)) +
        (SHR(q[i+3], 1) ^ SHL(q[i+3], 3) ^ ROTL64S(q[i+3],  4) ^ ROTL64B(q[i+3], 37)) +
        (SHR(q[i+4], 1) ^ SHL(q[i+4], 2) ^ ROTL64S(q[i+4], 13) ^ ROTL64B(q[i+4], 43)) +
        (SHR(q[i+5], 2) ^ SHL(q[i+5], 1) ^ ROTL64S(q[i+5], 19) ^ ROTL64B(q[i+5], 53)) +
        (SHR(q[i+6], 2) ^ SHL(q[i+6], 2) ^ ROTL64S(q[i+6], 28) ^ ROTL64B(q[i+6], 59)) +
        (SHR(q[i+7], 1) ^ SHL(q[i+7], 3) ^ ROTL64S(q[i+7],  4) ^ ROTL64B(q[i+7], 37)) +
        (SHR(q[i+8], 1) ^ SHL(q[i+8], 2) ^ ROTL64S(q[i+8], 13) ^ ROTL64B(q[i+8], 43)) +
        (SHR(q[i+9], 2) ^ SHL(q[i+9], 1) ^ ROTL64S(q[i+9], 19) ^ ROTL64B(q[i+9], 53)) +
        (SHR(q[i+10], 2) ^ SHL(q[i+10], 2) ^ ROTL64S(q[i+10], 28) ^ ROTL64B(q[i+10], 59)) +
        (SHR(q[i+11], 1) ^ SHL(q[i+11], 3) ^ ROTL64S(q[i+11],  4) ^ ROTL64B(q[i+11], 37)) +
        (SHR(q[i+12], 1) ^ SHL(q[i+12], 2) ^ ROTL64S(q[i+12], 13) ^ ROTL64B(q[i+12], 43)) +
        (SHR(q[i+13], 2) ^ SHL(q[i+13], 1) ^ ROTL64S(q[i+13], 19) ^ ROTL64B(q[i+13], 53)) +
        (SHR(q[i+14], 2) ^ SHL(q[i+14], 2) ^ ROTL64S(q[i+14], 28) ^ ROTL64B(q[i+14], 59)) +
        (SHR(q[i+15], 1) ^ SHL(q[i+15], 3) ^ ROTL64S(q[i+15],  4) ^ ROTL64B(q[i+15], 37)) +
        ((    d_x55[i] + ROTL64S(msg[i], i+1) +
            ROTL64S(msg[i+3], i+4) - ROTL64S(msg[i+10], i+11) ) ^ hash[i+7]);
    }

#pragma unroll 4
    for(int i=2;i<6;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    d_x55[i] + ROTL64S(msg[i], i+1) +
            ROTL64S(msg[i+3], i+4) - ROTL64S(msg[i+10], i+11) ) ^ hash[i+7]);
    }
#pragma unroll 3
    for(int i=6;i<9;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    d_x55[i] + ROTL64S(msg[i], i+1) +
            ROTL64S(msg[i+3], i+4) - ROTL64S(msg[i-6], i-5) ) ^ hash[i+7]);
    }
#pragma unroll 4
    for(int i=9;i<13;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    d_x55[i] + ROTL64S(msg[i], i+1) +
            ROTL64S(msg[i+3], i+4) - ROTL64S(msg[i-6], i-5) ) ^ hash[i-9]);
    }
#pragma unroll 3
    for(int i=13;i<16;i++) {
        q[i+16] = CONST_EXP2 + 
        ((    d_x55[i] + ROTL64S(msg[i], i+1) +
            ROTL64S(msg[i-13], i-12) - ROTL64S(msg[i-6], i-5) ) ^ hash[i-9]);
    }

    uint64_t XL64 = q[16]^q[17]^q[18]^q[19]^q[20]^q[21]^q[22]^q[23];
    uint64_t XH64 = XL64^q[24]^q[25]^q[26]^q[27]^q[28]^q[29]^q[30]^q[31];

    hash[0] =                       (SHL(XH64, 5) ^ SHR(q[16],5) ^ msg[ 0]) + (    XL64    ^ q[24] ^ q[ 0]);
    hash[1] =                       (SHR(XH64, 7) ^ SHL(q[17],8) ^ msg[ 1]) + (    XL64    ^ q[25] ^ q[ 1]);
    hash[2] =                       (SHR(XH64, 5) ^ SHL(q[18],5) ^ msg[ 2]) + (    XL64    ^ q[26] ^ q[ 2]);
    hash[3] =                       (SHR(XH64, 1) ^ SHL(q[19],5) ^ msg[ 3]) + (    XL64    ^ q[27] ^ q[ 3]);
    hash[4] =                       (SHR(XH64, 3) ^     q[20]    ^ msg[ 4]) + (    XL64    ^ q[28] ^ q[ 4]);
    hash[5] =                       (SHL(XH64, 6) ^ SHR(q[21],6) ^ msg[ 5]) + (    XL64    ^ q[29] ^ q[ 5]);
    hash[6] =                       (SHR(XH64, 4) ^ SHL(q[22],6) ^ msg[ 6]) + (    XL64    ^ q[30] ^ q[ 6]);
    hash[7] =                       (SHR(XH64,11) ^ SHL(q[23],2) ^ msg[ 7]) + (    XL64    ^ q[31] ^ q[ 7]);

    hash[ 8] = ROTL64S(hash[4], 9) + (    XH64     ^     q[24]    ^ msg[ 8]) + (SHL(XL64,8) ^ q[23] ^ q[ 8]);
    hash[ 9] = ROTL64S(hash[5],10) + (    XH64     ^     q[25]    ^ msg[ 9]) + (SHR(XL64,6) ^ q[16] ^ q[ 9]);
    hash[10] = ROTL64S(hash[6],11) + (    XH64     ^     q[26]    ^ msg[10]) + (SHL(XL64,6) ^ q[17] ^ q[10]);
    hash[11] = ROTL64S(hash[7],12) + (    XH64     ^     q[27]    ^ msg[11]) + (SHL(XL64,4) ^ q[18] ^ q[11]);
    hash[12] = ROTL64S(hash[0],13) + (    XH64     ^     q[28]    ^ msg[12]) + (SHR(XL64,3) ^ q[19] ^ q[12]);
    hash[13] = ROTL64S(hash[1],14) + (    XH64     ^     q[29]    ^ msg[13]) + (SHR(XL64,4) ^ q[20] ^ q[13]);
    hash[14] = ROTL64S(hash[2],15) + (    XH64     ^     q[30]    ^ msg[14]) + (SHR(XL64,7) ^ q[21] ^ q[14]);
    hash[15] = ROTL64S(hash[3],16) + (    XH64     ^     q[31]    ^ msg[15]) + (SHR(XL64,2) ^ q[22] ^ q[15]);
}

__global__ void quark_bmw512_gpu_hash_64(int threads, uint32_t startNounce, uint64_t *g_hash, uint32_t *g_nonceVector)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = (g_nonceVector != NULL) ? g_nonceVector[thread] : (startNounce + thread);

        int hashPosition = nounce - startNounce;
        uint64_t *inpHash = &g_hash[8 * hashPosition];
        uint64_t h[16];
        uint64_t message[16];

#pragma unroll 16
		for(int i=0;i<16;i++)
			h[i] = d_constMem[i];
#pragma unroll 8
        for(int i=0;i<8;i++)
            message[i] = inpHash[i];
#pragma unroll 6
        for(int i=9;i<15;i++)
            message[i] = 0;

        message[8] = SPH_C64(0x80);
        message[15] = SPH_C64(512);

        Compression512(message, h);

#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = d_final[i];

        Compression512(h, message);

        uint64_t *outpHash = &g_hash[8 * hashPosition];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = message[i+8];
    }
}

__global__ void quark_bmw512_gpu_hash_80(int threads, uint32_t startNounce, uint64_t *g_hash)
{
    int thread = (blockDim.x * blockIdx.x + threadIdx.x);
    if (thread < threads)
    {
        uint32_t nounce = startNounce + thread;

        // Init
        uint64_t h[16];
#pragma unroll 16
		for(int i=0;i<16;i++)
			h[i] = d_constMem[i];

        // Nachricht kopieren (Achtung, die Nachricht hat 64 Byte,
        // BMW arbeitet mit 128 Byte!!!
        uint64_t message[16];
#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = c_PaddedMessage80[i];

        // die Nounce durch die thread-spezifische ersetzen
        message[9] = REPLACE_HIWORD(message[9], cuda_swab32(nounce));

        // Compression 1
        Compression512(message, h);

        // Final
#pragma unroll 16
        for(int i=0;i<16;i++)
            message[i] = d_final[i];

        Compression512(h, message);

        // fertig
        uint64_t *outpHash = &g_hash[8 * thread];

#pragma unroll 8
        for(int i=0;i<8;i++)
            outpHash[i] = message[i+8];
    }
}

// Setup-Funktionen
__host__ void quark_bmw512_cpu_init(int thr_id, int threads)
{
    // nix zu tun ;-)
	// jetzt schon :D
	hipMemcpyToSymbol(HIP_SYMBOL(d_constMem), h_constMem, sizeof(h_constMem), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_x55), h_x55, sizeof(h_x55), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(d_final), h_final, sizeof(h_final), 0, hipMemcpyHostToDevice);
}

// Bmw512 f�r 80 Byte grosse Eingangsdaten
__host__ void quark_bmw512_cpu_setBlock_80(void *pdata)
{
	// Message mit Padding bereitstellen
	// lediglich die korrekte Nonce ist noch ab Byte 76 einzusetzen.
	unsigned char PaddedMessage[128];
	memcpy(PaddedMessage, pdata, 80);
	memset(PaddedMessage+80, 0, 48);
	uint64_t *message = (uint64_t*)PaddedMessage;
	// Padding einf�gen (Byteorder?!?)
	message[10] = SPH_C64(0x80);
	// L�nge (in Bits, d.h. 80 Byte * 8 = 640 Bits
	message[15] = SPH_C64(640);

	// die Message zur Berechnung auf der GPU
	hipMemcpyToSymbol( HIP_SYMBOL(c_PaddedMessage80), PaddedMessage, 16*sizeof(uint64_t), 0, hipMemcpyHostToDevice);
}

__host__ void quark_bmw512_cpu_hash_64(int thr_id, int threads, uint32_t startNounce, uint32_t *d_nonceVector, uint32_t *d_hash, int order)
{
    const int threadsperblock = TPB;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    quark_bmw512_gpu_hash_64<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash, d_nonceVector);

	hipDeviceSynchronize();
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__host__ void quark_bmw512_cpu_hash_80(int thr_id, int threads, uint32_t startNounce, uint32_t *d_hash, int order)
{
    const int threadsperblock = TPB;

    // berechne wie viele Thread Blocks wir brauchen
    dim3 grid((threads + threadsperblock-1)/threadsperblock);
    dim3 block(threadsperblock);

    // Gr��e des dynamischen Shared Memory Bereichs
    size_t shared_size = 0;

    quark_bmw512_gpu_hash_80<<<grid, block, shared_size>>>(threads, startNounce, (uint64_t*)d_hash);

	hipDeviceSynchronize();
}

#endif
